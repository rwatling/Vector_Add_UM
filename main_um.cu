#include "hip/hip_runtime.h"
// This program calculates the sum of two vectors using unifed memory
// By: Robbie Watling

# include "system_includes.h"

using namespace std;

// CUDA kernel for vector addition
__global__ void vectorAdd(int* a, int* b, int* c, int N) {
    // Calculate global thread thread ID
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

    // Boundary check
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    const int N = 1 << 4;
    size_t bytes = N * sizeof(int);

    // Declare unified memory pointers
    int* a, * b, * c;

    // Allocation memory for these pointers
    // Memory automatically managed
    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    hipMallocManaged(&c, bytes);

    // Initialize vectors
    for (int i = 0; i < N; i++) {
        a[i] = rand() % 100;
        b[i] = rand() % 100;
    }

    // Threads per CTA (1024 threads per CTA)
    int BLOCK_SIZE = 1 << 2;

    // CTAs per Grid
    int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Call CUDA kernel
    vectorAdd << <GRID_SIZE, BLOCK_SIZE >> > (a, b, c, N);

    // Wait for all previous operations before using values
    // We need this because we don't get the implicit synchronization of
    // hipMemcpy like in the original example
    hipDeviceSynchronize();

    // Verify the result on the CPU
    for (int i = 0; i < N; i++) {
        assert(c[i] == a[i] + b[i]);
    }

    // Free unified memory (same as memory allocated with hipMalloc)
    hipFree(a);
    hipFree(b);
    hipFree(c);

    cout << "COMPLETED SUCCESSFULLY!\n";

    return 0;
}